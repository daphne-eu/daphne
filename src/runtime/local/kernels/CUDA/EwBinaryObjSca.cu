#include "hip/hip_runtime.h"
/*
 * Copyright 2021 The DAPHNE Consortium
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "EwBinaryObjSca.h"

#include <runtime/local/datastructures/AllocationDescriptorCUDA.h>

#include "bin_ops.cuh"

template<class VT, class OP>
__global__ void ewBinMatSca(VT* res, const VT* lhs, const VT rhs, size_t N, OP op) {
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < N)
        res[tid] = op(lhs[tid], rhs);
}

namespace CUDA {
    template<typename VT>
    void EwBinaryObjSca<DenseMatrix<VT>, DenseMatrix<VT>, VT>::apply(BinaryOpCode opCode, DenseMatrix<VT> *&res,
            const DenseMatrix<VT> *lhs, VT rhs, DCTX(dctx)) {
        const size_t numRows = lhs->getNumRows();
        const size_t numCols = lhs->getNumCols();
    
        const size_t deviceID = 0; //ToDo: multi device support
        AllocationDescriptorCUDA alloc_desc(dctx, deviceID);
        
        int blockSize;
        int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
        size_t gridSize;

        if (res == nullptr)
            res = DataObjectFactory::create<DenseMatrix<VT>>(numRows, numCols, false, &alloc_desc);

        auto N = res->getNumItems();

        // ToDo: use templates instead of this if-else madness
        if (opCode == BinaryOpCode::ADD) {
            SumOp<VT> op;
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatSca<VT, SumOp<VT>>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;
            ewBinMatSca<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs, N, op);
        }
        else if (opCode == BinaryOpCode::MUL) {
            ProductOp<VT> op;
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatSca<VT, decltype(op)>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;
            ewBinMatSca<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs, N, op);
        }
        else if (opCode == BinaryOpCode::DIV) {
            DivOp<VT> op;
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatSca<VT, decltype(op)>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;
            ewBinMatSca<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs, N, op);
        }
        else if (opCode == BinaryOpCode::POW) {
            PowOp<VT> op;
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatSca<VT, decltype(op)>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;
            ewBinMatSca<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs, N, op);
        }
        else if (opCode == BinaryOpCode::SUB) {
            MinusOp<VT> op;
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatSca<VT, decltype(op)>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;
            ewBinMatSca<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs, N, op);
        }

        else {
            throw std::runtime_error(fmt::format("Unknown opCode {} for EwBinaryObjSca", static_cast<uint32_t>(opCode)));
        }
    }
    template struct EwBinaryObjSca<DenseMatrix<double>, DenseMatrix<double>, double>;
    template struct EwBinaryObjSca<DenseMatrix<float>, DenseMatrix<float>, float>;
    template struct EwBinaryObjSca<DenseMatrix<int64_t>, DenseMatrix<int64_t>, int64_t>;
    template struct EwBinaryObjSca<DenseMatrix<int32_t>, DenseMatrix<int32_t>, int32_t>;
    template struct EwBinaryObjSca<DenseMatrix<uint32_t>, DenseMatrix<uint32_t>, uint32_t>;
    template struct EwBinaryObjSca<DenseMatrix<uint64_t>, DenseMatrix<uint64_t>, uint64_t>;
}