#include "hip/hip_runtime.h"
/*
 * Copyright 2021 The DAPHNE Consortium
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "EwBinaryMat.h"
#include "HostUtils.h"
#include "runtime/local/datastructures/AllocationDescriptorCUDA.h"
#include "runtime/local/kernels/CUDA/bin_ops.cuh"

namespace CUDA {
    template<class VT, class OP>
    __global__ void ewBinMat(VT *res, const VT *lhs, const VT *rhs, size_t N, OP op) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto ltid = tid;
        if(ltid < N)
            res[ltid] = op(lhs[ltid], rhs[ltid]);
//        if(threadIdx.x < 1)
//            printf("bid=%d ltid=%d lhs=%4.3f rhs=%4.3f res=%4.3f\n", blockIdx.x, ltid, lhs[ltid], rhs[ltid], res[ltid]);
//	}
    }

// Todo: templatize this
    template<class VT, class OP>
    __global__ void ewBinMatRVec(VT *res, const VT *lhs, const VT *rhs, size_t dim, size_t N, OP op) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto ltid = tid;
//	while(ltid < N) {
        if(ltid < N) {
//        if(ltid == 9)
//            printf("C ltid=%d dim=%d ltid mod dim=%d\n", ltid, dim, ltid %dim);
            res[ltid] = op(lhs[ltid], rhs[ltid % dim]);
//		if(ltid == 9) {
//			printf("R ltid=%d ltidim=%d\n", ltid, ltid % dim);
//			printf("lhs[ltid]=%f\n",lhs[ltid]);
//			printf("rhs[ltid %% dim]=%f\n", rhs[ltid % dim]);
//			printf("res[ltid]=%f\n", res[ltid]);
//		}
//		ltid += gridDim.x;
        }
    }

    template<class VT, class OP>
    __global__ void ewBinMatCVec(VT *res, const VT *lhs, const VT *rhs, size_t dim, size_t N, OP op) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto ltid = tid;
//	while(ltid < N) {
        if(ltid < N) {
//		if(ltid == 9)
//			printf("C ltid=%d ltidim=%d\n", ltid, ltid/dim);
            res[ltid] = op(lhs[ltid], rhs[ltid / dim]);
//		ltid += gridDim.x;
        }
    }

    template<class VT, class OP>
    bool launch_ewbinmat(const size_t& numRowsLhs, const size_t& numColsLhs, const size_t& numRowsRhs,
            const size_t& numColsRhs, size_t& gridSize, int& minGridSize, int& blockSize, const size_t& N, VT* res, const VT* lhs,
            const VT* rhs) {
        OP op;

        if(numRowsLhs == numRowsRhs && numColsLhs == numColsRhs) {
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMat<VT, decltype(op)>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;
            ewBinMat<<<gridSize, blockSize>>>(res, lhs, rhs, N, op);
        }
        else if(numColsLhs == numColsRhs && (numRowsRhs == 1 || numRowsLhs == 1)) {
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatRVec<VT, decltype(op)>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;

            ewBinMatRVec<<<gridSize, blockSize>>>(res, lhs, rhs, numColsRhs, N, op);
        }
        else if(numRowsLhs == numRowsRhs && (numColsRhs == 1 || numColsLhs == 1)) {
            CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatCVec<VT, decltype(op)>, 0, 0));
            gridSize = (N + blockSize - 1) / blockSize;
            ewBinMatCVec<<<gridSize, blockSize>>>(res, lhs, rhs, numRowsRhs, N, op);
        }
        else
            return true;
        return false;
    }

// ----------------------------------------------------------------------------
// DenseMatrix <- DenseMatrix, DenseMatrix
// ----------------------------------------------------------------------------
    template<typename VTres, typename VTlhs, typename VTrhs>
    void EwBinaryMat<DenseMatrix<VTres>, DenseMatrix<VTlhs>, DenseMatrix<VTrhs>>::apply(BinaryOpCode opCode,
            DenseMatrix<VTres> *&res, const DenseMatrix<VTlhs> *lhs, const DenseMatrix<VTrhs> *rhs, DCTX(dctx)) {
        const size_t deviceID = 0; //ToDo: multi device support
        auto ctx = HIPContext::get(dctx, deviceID);
        AllocationDescriptorCUDA alloc_desc(dctx, deviceID);

        const size_t numRowsLhs = lhs->getNumRows();
        const size_t numColsLhs = lhs->getNumCols();
        const size_t numRowsRhs = rhs->getNumRows();
        const size_t numColsRhs = rhs->getNumCols();

        int blockSize = 0;
        int minGridSize = 0; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
        size_t gridSize = 0;

        if(res == nullptr)
            res = DataObjectFactory::create<DenseMatrix<VTres>>(numRowsLhs, numColsLhs, false, &alloc_desc);

        auto N = res->getNumItems();
        bool err = false;

        if(opCode == BinaryOpCode::ADD) {
            SumOp<VTres> op;
            if(numRowsLhs == numRowsRhs && numColsLhs == numColsRhs) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMat<VTres, SumOp<VTres>>, 0,
                                                           0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMat<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc), N,
                                                  op);
            }
            else if(numColsLhs == numColsRhs && (numRowsRhs == 1 || numRowsLhs == 1)) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatRVec<VTres, decltype(op)>,
                                                           0,
                                                           0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMatRVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc),
                                                      numColsRhs, N, op);
            }
            else if(numRowsLhs == numRowsRhs && (numColsRhs == 1 || numColsLhs == 1)) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatCVec<VTres, decltype(op)>,
                                                           0,
                                                           0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMatCVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc),
                                                      numRowsRhs, N, op);
            }
            else {
                err = true;
            }
        }
        else if(opCode == BinaryOpCode::SUB) {
            MinusOp<VTres> op;
            if(numRowsLhs == numRowsRhs && numColsLhs == numColsRhs) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMat<VTres, decltype(op)>, 0,
                                                           0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMat<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc), N,
                                                  op);
            }
            else if(numColsLhs == numColsRhs && (numRowsRhs == 1 || numRowsLhs == 1)) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatRVec<VTres, decltype(op)>,
                                                           0,
                                                           0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMatRVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc),
                                                      numColsRhs, N, op);
            }
            else if(numRowsLhs == numRowsRhs && (numColsRhs == 1 || numColsLhs == 1)) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatCVec<VTres, decltype(op)>,
                                                           0,
                                                           0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMatCVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc),
                                                      numRowsRhs, N, op);
            }
            else {
                err = true;
            }
        }
        else if(opCode == BinaryOpCode::MUL) {
            ProductOp<VTres> op;
            if(numRowsLhs == numRowsRhs && numColsLhs == numColsRhs) {
                CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMat<VTres, decltype(op)>,
                                                                0, 0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMat<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc),
                                                  rhs->getValues(&alloc_desc), N, op);
            }
            else if(numColsLhs == numColsRhs && (numRowsRhs == 1 || numRowsLhs == 1)) {
                CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatRVec<VTres,
                        decltype(op)>, 0, 0));
                gridSize = (N + blockSize - 1) / blockSize;

                ewBinMatRVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc),
                                                      rhs->getValues(&alloc_desc), numColsRhs, N, op);
            }
            else if(numRowsLhs == numRowsRhs && (numColsRhs == 1 || numColsLhs == 1)) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatCVec<VTres, decltype(op)>,
                                                           0, 0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMatCVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc),
                                                      rhs->getValues(&alloc_desc), numRowsRhs, N, op);
            }
            else {
                err = true;
            }
        }
        else if(opCode == BinaryOpCode::DIV) {
            DivOp<VTres> op;
            if(numRowsLhs == numRowsRhs && numColsLhs == numColsRhs) {
                CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMat<VTres, decltype(op)>,
                                                                0, 0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMat<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc),
                                                  rhs->getValues(&alloc_desc), N, op);
            }
            else if(numColsLhs == numColsRhs && (numRowsRhs == 1 || numRowsLhs == 1)) {
                CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatRVec<VTres,
                        decltype(op)>, 0, 0));
                gridSize = (N + blockSize - 1) / blockSize;

                ewBinMatRVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc),
                                                      rhs->getValues(&alloc_desc), numColsRhs, N, op);
            }
            else if(numRowsLhs == numRowsRhs && (numColsRhs == 1 || numColsLhs == 1)) {
                CHECK_CUDART(
                        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, ewBinMatCVec<VTres, decltype(op)>,
                                                           0, 0));
                gridSize = (N + blockSize - 1) / blockSize;
                ewBinMatCVec<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), lhs->getValues(&alloc_desc),
                                                      rhs->getValues(&alloc_desc), numRowsRhs, N, op);
            }
            else {
                err = true;
            }
        }

        else if (opCode == BinaryOpCode::MAX) {
            err = launch_ewbinmat<VTres, MaxOp<VTres>>(numRowsLhs, numColsLhs, numRowsRhs, numColsRhs, gridSize, minGridSize,
                blockSize, N, res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc));
        }
        else if (opCode == BinaryOpCode::NEQ) {
            err = launch_ewbinmat<VTres, NeqOp<VTres>>(numRowsLhs, numColsLhs, numRowsRhs, numColsRhs, gridSize, minGridSize,
                    blockSize, N, res->getValues(&alloc_desc), lhs->getValues(&alloc_desc), rhs->getValues(&alloc_desc));
        }
        else {
            throw std::runtime_error(fmt::format("Unknown opCode {} for EwBinaryMat", static_cast<uint32_t>(opCode)));
        }
        if(err) {
            throw std::runtime_error(
                            "EwBinaryMat (CUDA): "
                            "lhs and rhs must either have the same dimensions, "
                            "or one of them must be a row/column vector with the "
                            "width/height of the other"
            );
        }
        ctx->logger->debug("EwBinMat[{}]: {} blocks x {} threads = {} total threads for {} items",
                binary_op_codes[static_cast<int>(opCode)], gridSize, blockSize, gridSize*blockSize, N);
    }

    template struct EwBinaryMat<DenseMatrix<double>, DenseMatrix<double>, DenseMatrix<double>>;
    template struct EwBinaryMat<DenseMatrix<float>, DenseMatrix<float>, DenseMatrix<float>>;
    template struct EwBinaryMat<DenseMatrix<int64_t>, DenseMatrix<int64_t>, DenseMatrix<int64_t>>;
    template struct EwBinaryMat<DenseMatrix<int32_t>, DenseMatrix<int32_t>, DenseMatrix<int32_t>>;
    template struct EwBinaryMat<DenseMatrix<uint32_t>, DenseMatrix<uint32_t>, DenseMatrix<uint32_t>>;
    template struct EwBinaryMat<DenseMatrix<uint64_t>, DenseMatrix<uint64_t>, DenseMatrix<uint64_t>>;
}
