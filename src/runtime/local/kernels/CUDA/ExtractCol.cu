#include "hip/hip_runtime.h"
/*
 * Copyright 2021 The DAPHNE Consortium
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ExtractCol.h"

#include <runtime/local/datastructures/AllocationDescriptorCUDA.h>

namespace CUDA {
    template<class DTRes, class DTArg, class DTSel>
    __global__ void extract_col(DTRes *res, const DTArg *arg, const DTSel *sel, const size_t sel_rows, const size_t arg_cols, const size_t cols) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        if(tid < cols) {
            auto idx = sel[tid%sel_rows];
            auto row = tid / sel_rows;
            res[tid] = arg[row * arg_cols + idx];
        }
    }

    // ----------------------------------------------------------------------------
    // DenseMatrix <- DenseMatrix, DenseMatrix
    // ----------------------------------------------------------------------------
    template<class DTRes, class DTArg, class DTSel>
    void ExtractCol<DenseMatrix<DTRes>, DenseMatrix<DTArg>, DenseMatrix<DTSel>>::apply(DenseMatrix<DTRes>*& res,
            const DenseMatrix<DTArg>* arg, const DenseMatrix<DTSel>* sel, DCTX(dctx)) {
        const size_t deviceID = 0; //ToDo: multi device support
        AllocationDescriptorCUDA alloc_desc(dctx, deviceID);
        if(res == nullptr) {
            res = DataObjectFactory::create<DenseMatrix<DTRes>>(arg->getNumRows(), sel->getNumRows(), false,
                    &alloc_desc);
        }
        
        auto N = res->getNumItems();
        int blockSize;
        int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch
        size_t gridSize;
        CHECK_CUDART(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, extract_col<DTRes, DTArg, DTSel>, 0, 0));
        gridSize = (N + blockSize - 1) / blockSize;

        spdlog::get("runtime::cuda")->debug("ExtractCol: {} blocks x {} threads = {} total threads for {} items",
                gridSize, blockSize, gridSize*blockSize, N);

        extract_col<<<gridSize, blockSize>>>(res->getValues(&alloc_desc), arg->getValues(&alloc_desc),
                sel->getValues(&alloc_desc), sel->getNumRows(), arg->getNumCols(), N);
    }
    template struct ExtractCol<DenseMatrix<int64_t>, DenseMatrix<int64_t>, DenseMatrix<int64_t>>;
    template struct ExtractCol<DenseMatrix<float>, DenseMatrix<float>, DenseMatrix<int64_t>>;
    template struct ExtractCol<DenseMatrix<double>, DenseMatrix<double>, DenseMatrix<int64_t>>;
}